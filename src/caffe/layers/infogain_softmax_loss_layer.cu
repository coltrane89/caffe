#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void InfogainSoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const Dtype* infogain_mat, const int Hdim,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN))) * infogain_mat[label_value * Hdim + label_value];
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void InfogainSoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const Dtype* infogain_mat = NULL;
  if (bottom.size() < 3) {
    infogain_mat = infogain_.gpu_data();
  } else {
    infogain_mat = bottom[2]->gpu_data();
  }
  const int dim = prob_.count() / outer_num_;
  const int Hdim = dim / inner_num_;
  const int nthreads = outer_num_ * inner_num_;
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  InfogainSoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data, infogain_mat,
      Hdim, outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  if (normalize_) {
    Dtype count;
    caffe_gpu_asum(nthreads, counts, &count);
    loss /= count;
  } else {
    loss /= outer_num_;
  }
  top[0]->mutable_cpu_data()[0] = loss;
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void InfogainSoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, const Dtype* infogain_mat, Dtype* bottom_diff, 
          const int Hdim, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= infogain_mat[label_value * Hdim + label_value];
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void InfogainSoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down.size() > 2 && propagate_down[2]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to infogain inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* infogain_mat = NULL;   
    if (bottom.size() < 3) {
      infogain_mat = infogain_.gpu_data();
    } else {
      infogain_mat = bottom[2]->gpu_data();
    }
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int Hdim = dim / inner_num_;
    const int nthreads = outer_num_ * inner_num_;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
    InfogainSoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, infogain_mat, bottom_diff,
        Hdim, outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    if (normalize_) {
      Dtype count;
      caffe_gpu_asum(nthreads, counts, &count);
      caffe_gpu_scal(prob_.count(), loss_weight / count, bottom_diff);
    } else {
      caffe_gpu_scal(prob_.count(), loss_weight / outer_num_, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(InfogainSoftmaxWithLossLayer);

}  // namespace caffe
